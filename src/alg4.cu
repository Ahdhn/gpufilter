#include "hip/hip_runtime.h"
/**
 *  @file alg4.cu
 *  @brief CUDA device code for GPU-Efficient Recursive Filtering Algorithm 4
 *  @author Rodolfo Lima
 *  @date September, 2011
 */

//== INCLUDES =================================================================

#include <cmath>
#include <cstdio>
#include <cfloat>
#include <cassert>
#include <iostream>
#include <algorithm>

#include <util.h>
#include <symbol.h>
#include <dvector.h>

#include <gpudefs.h>
#include <gpufilter.h>
#include <gpuconsts.cuh>

#include <alg4.cuh>

//== NAMESPACES ===============================================================

namespace gpufilter {

//== IMPLEMENTATION ===========================================================

//-- Device -------------------------------------------------------------------

__device__ float2 operator + ( const float2 &a,
                               const float2 &b ) {
    return make_float2(a.x+b.x, a.y+b.y);
}


__device__ float2& operator += ( float2& a,
                                 const float2& b ) {
    a.x += b.x;
    a.y += b.y;
    return a;
}


__device__ float2 operator * ( const float2& a,
                               float b ) {
    return make_float2(a.x*b, a.y*b);
}


__device__ float2 operator * ( float a,
                               const float2& b ) {
    return b*a;
}


__device__ float2 operator / ( const float2& a,
                               float b ) {
    return make_float2(a.x/b, a.y/b);
}


__device__ float2 mul2x2( const float2& v,
                          float mat[2][2] ) {
    return make_float2(v.x*mat[0][0] + v.y*mat[1][0],
                       v.x*mat[0][1] + v.y*mat[1][1]);
}

//-- Algorithm 4_2 Stage 1 ----------------------------------------------------

__global__ __launch_bounds__(WS*DW, DNB)
void alg4_stage1( float2 *g_transp_pybar,
                  float2 *g_transp_ezhat )
{
    int tx = threadIdx.x, ty = threadIdx.y, m = blockIdx.x*2, n = blockIdx.y;

    // Each cuda block will work on two horizontally adjacent WSxWS
    // input data blocks, so allocate enough shared memory for these.
    __shared__ float s_block[WS*2][WS+1];

    float (*bdata)[WS+1] = (float (*)[WS+1]) &s_block[ty][tx],
          (*bdata2)[WS+1] = (float (*)[WS+1])&s_block[ty+WS][tx];

    // Load data into shared memory
    float tu = ((m-c_border)*WS+tx+.5f)*c_inv_width,
          tv = ((n-c_border)*WS+ty+.5f)*c_inv_height;

#pragma unroll
    for(int i=0; i<WS-(WS%DW); i+=DW)
    {
        **bdata = tex2D(t_in, tu, tv);
        bdata += DW;

        **bdata2 = tex2D(t_in, tu+WS*c_inv_width, tv);
        bdata2 += DW;

        tv += DW*c_inv_height;
    }

    if(ty < WS%DW)
    {
        **bdata = tex2D(t_in, tu, tv);
        **bdata2 = tex2D(t_in, tu+WS*c_inv_width, tv);
    }

    //if(ty < 2)
        m += ty;

    if(m >= c_m_size)
        return;

    // We use a transposed matrix for pybar and ezhat to have
    // coalesced memory accesses. This is the index for these
    // transposed buffers.
    g_transp_pybar += m*c_carry_height + n*WS + tx; 
    g_transp_ezhat += m*c_carry_height + n*WS + tx;

    __syncthreads();

    float2 prev; // .x -> p0, .y -> p1

    if(ty < 2)
    {
        float *bdata = s_block[tx+ty*WS];

        // calculate pybar, scan left -> right
        prev = make_float2(0,*bdata++);

#pragma unroll
        for(int j=1; j<WS; ++j, ++bdata)
        {
            *bdata = prev.x = *bdata - prev.y*c_a1 - prev.x*c_a2;

            swap(prev.x, prev.y);
        }

        if(m < c_m_size-1)
            *g_transp_pybar = prev*c_b0;


        if(m > 0)
        {
            // calculate ezhat, scan right -> left
            prev = make_float2(*--bdata, 0);

            --bdata;

#pragma unroll
            for(int j=WS-2; j>=0; --j, --bdata)
            {
                *bdata = prev.y = *bdata - prev.x*c_a1 - prev.y*c_a2;
                swap(prev.x, prev.y);
            }

            *g_transp_ezhat = prev*(c_b0*c_b0);
        }
    }
}

//-- Algorithm 4_2 Stage 2 and 3 or Stage 5 and 6 -----------------------------

__global__ __launch_bounds__(WS*DW, DNB)
void alg4_stage2_3( float2 *g_transp_pybar,
                    float2 *g_transp_ezhat )
{
    int tx = threadIdx.x, ty = threadIdx.y, n = blockIdx.y;

    __shared__ float2 s_transp_block[DW][WS];
    float2 *bdata = &s_transp_block[ty][tx];

    // P(ybar) -> P(y) processing --------------------------------------

    float2 *transp_pybar = g_transp_pybar + ty*c_carry_height + n*WS+tx;

    // first column-block

    // read P(ybar)
    *bdata = *transp_pybar;

    float2 py; // P(Y), .x = p0, .y = p1

    __syncthreads();

    if(ty == 0)
    {
        float2 (*bdata)[WS] = (float2 (*)[WS]) &s_transp_block[0][tx];

        // (24): P_m(y) = P_m(ybar) + A^b_F * P_{m-1}(y)
        py = **bdata++;

#pragma unroll
        for(int m=1; m<blockDim.y; ++m, ++bdata)
            **bdata = py = **bdata + mul2x2(py,c_AbF2);
    }
    

    __syncthreads();

    // write P(y)
    if(ty > 0) // first one doesn't need fixing
        *transp_pybar = *bdata;

    transp_pybar += c_carry_height*blockDim.y;

    // middle column-blocks
    int m = blockDim.y;
    if(m == DW)
    {
        int mmax = c_m_size-(c_m_size%DW)-1;
        for(; m<mmax; m+=DW)
        {
            *bdata = *transp_pybar;

            __syncthreads();

            if(ty == 0)
            {
                float2 (*bdata)[WS] = (float2 (*)[WS]) &s_transp_block[0][tx];
#pragma unroll
                for(int dm=0; dm<DW; ++dm, ++bdata)
                    **bdata = py = **bdata + mul2x2(py,c_AbF2);
            }

            __syncthreads();

            *transp_pybar = *bdata;
            transp_pybar += c_carry_height*DW;
        }
    }

    // remaining column-blocks
    if(m < c_m_size-1)
    {
        if(m+ty < c_m_size-1)
            *bdata = *transp_pybar;

        int remaining = c_m_size-1 - m;

        __syncthreads();

        if(ty == 0)
        {
            float2 (*bdata)[WS] = (float2 (*)[WS]) &s_transp_block[0][tx];
#pragma unroll
            for(int dm=0; dm<remaining; ++dm, ++bdata)
                **bdata = py = **bdata + mul2x2(py,c_AbF2);

        }

        __syncthreads();

        if(m+ty < c_m_size-1)
            *transp_pybar = *bdata;
    }

    // E(zhat) -> E(z) processing --------------------------------------

    int idx = (c_m_size-1-ty)*c_carry_height + n*WS+tx;

    const float2 *transp_pm1y = g_transp_pybar + idx - c_carry_height;

    // last column-block
    float2 *transp_ezhat = g_transp_ezhat + idx;

    m = c_m_size-1;

    // all pybars must be updated!
    __syncthreads();

    float2 ez;

    if(m-ty > 0)
    {
        *bdata = *transp_ezhat;

        *bdata += mul2x2(*transp_pm1y,c_AFP_HARB);

        __syncthreads();

        if(ty == 0)
        {
            float2 (*bdata)[WS] = (float2 (*)[WS]) &s_transp_block[0][tx];
            ez = **bdata++;

            for(int dm=1; dm<blockDim.y; ++dm, ++bdata)
                **bdata = ez = **bdata + mul2x2(ez,c_AbR2);
        }

        __syncthreads();

        *transp_ezhat = *bdata;

    }
    transp_ezhat -= c_carry_height*blockDim.y;
    transp_pm1y -= c_carry_height*blockDim.y;

    // middle column-blocks
    m = c_m_size-1 - blockDim.y;
    if(blockDim.y == DW)
    {
        int mmin = c_m_size%DW;
        for(; m>=mmin; m-=DW)
        {
            if(m > 0)
            {
                *bdata = *transp_ezhat;

                if(m-ty > 0)
                    *bdata += mul2x2(*transp_pm1y,c_AFP_HARB);

                __syncthreads();

                if(ty == 0)
                {
                    float2 (*bdata)[WS] = (float2 (*)[WS]) &s_transp_block[0][tx];
#pragma unroll
                    for(int dm=0; dm<DW; ++dm, ++bdata)
                        **bdata = ez = **bdata + mul2x2(ez,c_AbR2);
                }

                __syncthreads();

                *transp_ezhat = *bdata;
            }

            transp_ezhat -= DW*c_carry_height;
            transp_pm1y -= DW*c_carry_height;
        }
    }

    // remaining column-blocks
    if(m > 0)
    {
        int remaining = m+1;

        if(m-ty >= 0)
        {
            *bdata = *transp_ezhat;
        
            if(m-ty > 0)
                *bdata += mul2x2(*transp_pm1y,c_AFP_HARB);
        }

        __syncthreads();

        if(ty == 0)
        {
            float2 (*bdata)[WS] = (float2 (*)[WS]) &s_transp_block[0][tx];
            // (24): P_m(y) = P_m(ybar) + A^b_F * P_{m-1}(y)
#pragma unroll
            for(int dm=0; dm<remaining; ++dm, ++bdata)
                **bdata = ez = **bdata + mul2x2(ez,c_AbR2);
        }

        __syncthreads();

        if(m-ty > 0)
            *transp_ezhat = *bdata;
    }
}

//-- Algorithm 4_2 Stage 4 or Stage 7 -----------------------------------------

template <bool FUSION>
__global__ __launch_bounds__(WS*DW, DNB)
void alg4_stage4( float *g_transp_out,
                  float2 *g_transp_py,
                  float2 *g_transp_ez,
                  float2 *g_pubar,
                  float2 *g_evhat )
{
    int tx = threadIdx.x, ty = threadIdx.y, m = blockIdx.x*2, n = blockIdx.y;

    // Each cuda block will work on two horizontally adjacent WSxWS
    // input data blocks, so allocate enough shared memory for these.
    __shared__ float s_block[WS*2][WS+1];

    float (*bdata)[WS+1] = (float (*)[WS+1]) &s_block[ty][tx],
          (*bdata2)[WS+1] = (float (*)[WS+1])&s_block[ty+WS][tx];

    // Load data into shared memory
    float tu = ((m-c_border)*WS+tx + 0.5f)*c_inv_width,
          tv = ((n-c_border)*WS+ty + 0.5f)*c_inv_height;

#pragma unroll
    for(int i=0; i<WS-(WS%DW); i+=DW)
    {
        **bdata = tex2D(t_in, tu, tv);
        bdata += DW;

        **bdata2 = tex2D(t_in, tu+WS*c_inv_width, tv);
        bdata2 += DW;

        tv += DW*c_inv_height;
    }

    if(ty < WS%DW)
    {
        **bdata = tex2D(t_in, tu, tv);
        **bdata2 = tex2D(t_in, tu+WS*c_inv_width, tv);
    }

    m += ty;

    if(m >= c_m_size)
        return;

    // We use a transposed matrix for py and ez to have coalesced
    // memory accesses. This is the index for these transposed
    // buffers.
    g_transp_py += (m-1)*c_carry_height + n*WS + tx; 
    g_transp_ez += (m+1)*c_carry_height + n*WS + tx;

    __syncthreads();

    if(ty < 2)
    {
        float2 prev; // .x -> p0, .y -> p1

        float *bdata = s_block[tx+ty*WS];

        // calculate pybar, scan left -> right
        if(m > 0)
            prev = *g_transp_py * c_inv_b0;
        else
            prev = make_float2(0,0);

#pragma unroll
        for(int j=0; j<WS; ++j, ++bdata)
        {
            *bdata = prev.x = *bdata - prev.y*c_a1 - prev.x*c_a2;

            swap(prev.x, prev.y);
        }
        --bdata;

        // calculate ez, scan right -> left
        if(m < c_m_size-1)
            prev = *g_transp_ez;
        else
            prev = make_float2(0,0);

        float b0_2 = c_b0*c_b0;

        // For some reason it's faster when this is here then inside
        // the next if block
        int x = (m-c_border+1)*WS-1;
        int y = (n-c_border)*WS+tx;

        // current block intersects transp_out's area?
        if(m >= c_border && m <= c_last_m && n >= c_border && n <= c_last_n)
        {
            // image's end is in the middle of the block and we're outside
            // the image width?
            if(x >= c_width)
            {
                // process data until we get into the image
                int j;
#pragma unroll
                for(j=x; j>=c_width; --j, --bdata)
                {
                    prev.y = *bdata*b0_2 - prev.x*c_a1 - prev.y*c_a2;

                    if(FUSION)
                        *bdata = prev.y;

                    swap(prev.x, prev.y);
                }

                // now we're inside the image, we must write to transp_out
                float *out = g_transp_out + (c_width-1)*c_transp_out_height + y;

                int mmin = x-(WS-1);

#pragma unroll
                for(;j>=mmin; --j, --bdata, out -= c_transp_out_height)
                {
                    prev.y = *bdata*b0_2 - prev.x*c_a1 - prev.y*c_a2;

                    if(FUSION)
                        *bdata = prev.y;

                    if(y < c_height)
                        *out = prev.y;

                    swap(prev.x, prev.y);
                }
            }
            else
            {
                float *out = g_transp_out + x*c_transp_out_height + y;

#pragma unroll
                for(int j=WS-1; j>=0; --j, --bdata, out -= c_transp_out_height)
                {
                    prev.y = *bdata*b0_2 - prev.x*c_a1 - prev.y*c_a2;

                    if(FUSION)
                        *bdata = prev.y;

                    if(y < c_height)
                        *out = prev.y;
                    swap(prev.x, prev.y);
                }
            }
        }
        else
        {
#pragma unroll
            for(int j=WS-1; j>=0; --j, --bdata)
            {
                prev.y = *bdata*b0_2 - prev.x*c_a1 - prev.y*c_a2;

                if(FUSION)
                    *bdata = prev.y;

                swap(prev.x, prev.y);
            }
        }

        if(FUSION)
        {
            g_pubar += n*c_carry_width + m*WS + tx; 
            g_evhat += n*c_carry_width + m*WS + tx;

            float (*bdata)[WS+1] = (float (*)[WS+1]) &s_block[ty*WS][tx];

            // calculate pubar, scan left -> right
            float2 prev = make_float2(0,**bdata++);

#pragma unroll
            for(int i=1; i<WS; ++i, ++bdata)
            {
                **bdata = prev.x = **bdata - prev.y*c_a1 - prev.x*c_a2;

                swap(prev.x, prev.y);
            }

            if(n < c_n_size-1)
                *g_pubar = prev*c_b0;

            if(n > 0)
            {
                // calculate evhat, scan right -> left
                prev = make_float2(**--bdata, 0);

                --bdata;

#pragma unroll
                for(int i=WS-2; i>=0; --i, --bdata)
                {
                    prev.y = **bdata - prev.x*c_a1 - prev.y*c_a2;
                    swap(prev.x, prev.y);
                }

                *g_evhat = prev*b0_2;
            }
        }
    }
}

//-- Host ---------------------------------------------------------------------

__host__
void prepare_alg4( dvector<float>& d_out,
                   dvector<float>& d_transp_out,
                   int& transp_out_height,
                   hipArray *& a_in,
                   dvector<float2>& d_transp_pybar,
                   dvector<float2>& d_transp_ezhat,
                   dvector<float2>& d_pubar,
                   dvector<float2>& d_evhat,
                   dim3& cg_img,
                   const float *h_in,
                   const int& h,
                   const int& w,
                   const float& b0,
                   const float& a1,
                   const float& a2,
                   const initcond& ic,
                   const int& extb )
{

    up_constants_coefficients2( b0, a1, a2 );

    // cuda channel descriptor for texture
    hipChannelFormatDesc ccd = hipCreateChannelDesc<float>();
    hipMallocArray( &a_in, &ccd, w, h );
    hipMemcpyToArray( a_in, 0, 0, h_in, w*h*sizeof(float),
                       hipMemcpyHostToDevice );

    d_out.resize( w * h );

    // hipBindTexture2D chokes when memory block stride isn't
    // multiple of 256 bytes, let's add some padding.
    transp_out_height = ((h+WS-1)/WS)*WS;
    d_transp_out.resize( w * transp_out_height );

    copy_to_symbol("c_transp_out_height", transp_out_height);

    int ext_h, ext_w;
    up_constants_sizes( cg_img, ext_h, ext_w, h, w, extb );

    d_transp_pybar.resize( cg_img.x * ext_h );
    d_transp_ezhat.resize( cg_img.x * ext_h );
    d_pubar.resize( cg_img.y * ext_w );
    d_evhat.resize( cg_img.y * ext_w );

    d_transp_pybar.fill_zero();
    d_transp_ezhat.fill_zero();
    d_pubar.fill_zero();
    d_evhat.fill_zero();

    t_in.normalized = true;
    t_in.filterMode = hipFilterModePoint;

    switch( ic ) {
    case zero: // mode border defaults to zero-border
        t_in.addressMode[0] = t_in.addressMode[1] = hipAddressModeBorder;
        break;
    case clamp:
        t_in.addressMode[0] = t_in.addressMode[1] = hipAddressModeClamp;
        break;
    case repeat: // mode wrap implements repeat
        t_in.addressMode[0] = t_in.addressMode[1] = hipAddressModeWrap;
        break;
    case mirror:
        t_in.addressMode[0] = t_in.addressMode[1] = hipAddressModeMirror;
        break;
    }

}

__host__
void alg4( dvector<float>& d_out,
           dvector<float>& d_transp_out,
           int& transp_out_height,
           const int& h,
           const int& w,
           const hipArray *a_in,
           dvector<float2>& d_transp_pybar,
           dvector<float2>& d_transp_ezhat,
           dvector<float2>& d_pubar,
           dvector<float2>& d_evhat,
           const dim3& cg_img )
{

    dvector<float2> d_transp_py, d_transp_ez, d_pu, d_ev;

    hipBindTextureToArray( t_in, a_in );

    alg4_stage1<<<
        dim3((cg_img.x+2-1)/2, cg_img.y), dim3(WS, DW) >>>(
            d_transp_pybar, d_transp_ezhat );

    alg4_stage2_3<<<
        dim3(1, cg_img.y), dim3(WS, std::min<int>(cg_img.x, DW)) >>>(
            d_transp_pybar, d_transp_ezhat );

    swap( d_transp_pybar, d_transp_py );
    swap( d_transp_ezhat, d_transp_ez );

    alg4_stage4<true><<<
        dim3((cg_img.x+2-1)/2, cg_img.y), dim3(WS, DW) >>>(
            d_transp_out, d_transp_py, d_transp_ez, d_pubar, d_evhat );

    alg4_stage2_3<<<
        dim3(1, cg_img.x), dim3(WS, std::min<int>(cg_img.y, DW)) >>>(
            d_pubar, d_evhat );

    swap( d_pubar, d_pu );
    swap( d_evhat, d_ev );

    hipUnbindTexture( t_in );

    size_t offset;
    hipBindTexture2D( &offset, t_in, d_transp_out, h, w,
                       transp_out_height*sizeof(float) );

    alg4_stage4<false><<<
        dim3((cg_img.y+2-1)/2, cg_img.x), dim3(WS, DW) >>>(
            d_out, d_pu, d_ev, 0, 0 );

    swap( d_ev, d_evhat );
    swap( d_pu, d_pubar );
    swap( d_transp_ez, d_transp_ezhat );
    swap( d_transp_py, d_transp_pybar );

    hipUnbindTexture( t_in );

}

__host__
void alg4( float *h_inout,
           const int& h,
           const int& w,
           const float& b0,
           const float& a1,
           const float& a2,
           const initcond& ic,
           const int& extb )
{

    dim3 cg_img;
    dvector<float> d_out, d_transp_out;
    dvector<float2> d_transp_pybar, d_transp_ezhat, d_pubar, d_evhat;
    int transp_out_height;
    hipArray *a_in;

    prepare_alg4( d_out, d_transp_out, transp_out_height, a_in,
                  d_transp_pybar, d_transp_ezhat, d_pubar, d_evhat,
                  cg_img, h_inout, h, w, b0, a1, a2, ic, extb );

    alg4( d_out, d_transp_out, transp_out_height, h, w, a_in,
          d_transp_pybar, d_transp_ezhat, d_pubar, d_evhat, cg_img );

    d_out.copy_to( h_inout, w * h );

    hipFreeArray( a_in );

}

//=============================================================================
} // namespace gpufilter
//=============================================================================
// vi: ai ts=4 sw=4
