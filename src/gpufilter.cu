#include "hip/hip_runtime.h"
/**
 *  @file gpufilter.cu
 *  @brief CUDA device code for GPU-Efficient Recursive Filtering
 *  @author Rodolfo Lima
 *  @date September, 2011
 */

//== INCLUDES =================================================================

#include <cmath>
#include <cstdio>
#include <cfloat>
#include <cassert>
#include <iostream>
#include <algorithm>

#include <timer.h>
#include <symbol.h>
#include <dvector.h>

#include <gpufilter.h>
#include <gpudefs.cuh>
#include <gpuconsts.cuh>

#include <gpufilter.cuh>

//== NAMESPACES ===============================================================

namespace gpufilter {

//== IMPLEMENTATION ===========================================================

//-- Device -------------------------------------------------------------------

template <class T> 
__device__
inline void swap(T& a, T& b) {
    T c = a;
    a = b;
    b = c;
}

__device__ inline
float2 multgpu(const float M[2][2], const float2& v)
{
    return make_float2(M[0][0]*v.x + M[0][1]*v.y, M[1][0]*v.x + M[1][1]*v.y);
}

__device__ inline
float2 addgpu(const float2& u, const float2& v)
{
    return make_float2(u.x+v.x, u.y+v.y);
}

__device__ inline
float2 addgpu(const float2& u, const float2& v, const float2& w)
{
    return make_float2(u.x+v.x+w.x, u.y+v.y+w.y);
}

//-- Algorithm 4_2 ------------------------------------------------------------

__global__ __launch_bounds__(WS*SOW, MBO)
void alg4_stage1( const float *g_in,
                  float2 *g_transp_ybar,
                  float2 *g_transp_zhat )
{
    int m = blockIdx.y, n = blockIdx.x;
    int tx = threadIdx.x, ty = threadIdx.y;

    __shared__ float block[WS][WS+1];

    g_in += (m*WS + ty)*c_width + n*WS + tx;

#pragma unroll
    for(int i=0; i<WS-2; i+=SOW)
    {
        block[ty+i][tx] = g_in[i*c_width];
    }

    if(ty < 2)
    {
        block[ty+WS-2][tx] = g_in[(WS-2)*c_width];
    }

    __syncthreads();

    if(ty == 0) // one warp computing
    {
        float *bdata = block[tx];
        int outidx = n*c_height + m*WS + tx; // transposed!

        float2 accum;

        if(n < c_n_size-1)
        {
            accum.x = bdata[0];
            accum.y = bdata[1] -= c_Minf*accum.x;

            for(int i=2; i<WS; ++i)
            {
                accum.x = bdata[i] -= c_Minf*accum.y + c_Linf2*accum.x;
                swap(accum.x, accum.y);
            }
        }
        else
        {
            accum.x = bdata[0];
            accum.y = bdata[1] -= c_Minf*accum.x;

            for(int i=2; i<WS-1; ++i)
            {
                accum.x = bdata[i] -= c_Minf*accum.y + c_Linf2*accum.x;
                swap(accum.x, accum.y);
            }
            accum.x = bdata[WS-1] -= c_Minf*accum.y + c_Linf2*accum.x;
            swap(accum.x, accum.y);
        }

        g_transp_ybar[outidx] = accum;

        if(n < c_n_size-1)
        {
            accum.y = bdata[WS-1] * c_Linf2;
            accum.x = (bdata[WS-2] - accum.y*c_Ninf)*c_Linf2;

#pragma unroll
            for(int i=WS-3; i>=0; --i)
            {
                accum.y = (bdata[i] - accum.x*c_Ninf - accum.y)*c_Linf2;
                swap(accum.x, accum.y);
            }
        }
        else // last block
        {
            int i = WS-1;

            accum.y = bdata[i--] * c_Llast2;
            accum.x = (bdata[i--] - accum.y*c_Ninf)*c_Linf2;

            for(; i>=0; --i)
            {
                accum.y = (bdata[i] - accum.x*c_Ninf - accum.y)*c_Linf2;
                swap(accum.x, accum.y);
            }

        }

        g_transp_zhat[outidx] = accum;
    }
}

__global__ __launch_bounds__(MTS, MBO)
void alg4_stage2_3_or_5_6( float2 *g_transp_ybar,
                           float2 *g_transp_zhat )
{
    int m = blockIdx.x;
    int tx = threadIdx.x;

    int row = m*blockDim.x + tx;

    if(row >= c_height) return;

    g_transp_ybar += row;
    g_transp_zhat += row;

    float2 accum = g_transp_ybar[0];

    for(int j=1; j<c_n_size; ++j)
    {
        g_transp_ybar += c_height;

        *g_transp_ybar = accum = addgpu(*g_transp_ybar,
                                        multgpu(c_Af, accum));
    }

    g_transp_zhat += (c_n_size-1)*c_height;
    g_transp_ybar -= c_height;

    *g_transp_zhat = accum = addgpu(*g_transp_zhat,
                                    multgpu(c_Arf, *g_transp_ybar));

    for(int j=c_n_size-2; j>=1; --j)
    {
        g_transp_ybar -= c_height;
        g_transp_zhat -= c_height;

        *g_transp_zhat = accum = addgpu(*g_transp_zhat,
                                        multgpu(c_Ar, accum),
                                        multgpu(c_Arf, *g_transp_ybar));
    }
}

__global__ __launch_bounds__(WS*SOW, ONB)
void alg4_stage4( float *g_inout,
                  const float2 *g_transp_y,
                  const float2 *g_transp_z,
                  float2 *g_ubar,
                  float2 *g_vhat )
{
    int m = blockIdx.y*2, n = blockIdx.x;
    int tx = threadIdx.x, ty = threadIdx.y;

    __shared__ float block[WS*2][WS+1];

    g_inout += (m*WS + ty)*c_width + n*WS + tx;

#pragma unroll
    for(int i=0; i<WS-2; i+=SOW)
    {
        block[ty+i][tx] = g_inout[i*c_width];
        block[ty+i+WS][tx] = g_inout[(i+WS)*c_width];
    }

    if(ty < 2)
    {
        block[ty+WS-2][tx] = g_inout[(WS-2)*c_width];
        block[ty+WS-2+WS][tx] = g_inout[(WS-2+WS)*c_width];
    }

    __syncthreads();

    if(ty < 2)
    {
        m += ty;

        int outidx = n*c_height + m*WS + tx; // transposed!

        float *bdata = block[tx+ty*WS];

        if(n < c_n_size-1)
        {
            float2 accum;
            
            if(n == 0)
            {
                accum.x = bdata[0];
                accum.y = bdata[1] -= c_Minf*accum.x;
            }
            else
                accum = g_transp_y[outidx-c_height];

#pragma unroll
            for(int i=0; i<WS; ++i)
            {
                accum.x = bdata[i] -= c_Minf*accum.y + c_Linf2*accum.x;
                swap(accum.x, accum.y);
            }
        }
        else
        {
            float2 accum = g_transp_y[outidx-c_height];

            for(int i=0; i<WS-1; ++i)
            {
                accum.x = bdata[i] -= c_Minf*accum.y + c_Linf2*accum.x;
                swap(accum.x, accum.y);
            }
            bdata[WS-1] -= c_Minf*accum.y + c_Linf2*accum.x;

        }

        if(n < c_n_size-1)
        {
            float2 accum = g_transp_z[outidx+c_height];

#pragma unroll
            for(int i=WS-1; i>=0; --i)
            {
                accum.y = bdata[i] = (bdata[i] - accum.x*c_Ninf - accum.y)*c_Linf2;
                swap(accum.x,accum.y);
            }
        }
        // last block
        else
        {
            float2 accum;
            int i = WS-1;

            accum.y = bdata[i--] *= c_Linf2;
            accum.x = bdata[i] = (bdata[i] - accum.y*c_Ninf)*c_Linf2;
            --i;

            for(; i>=0; --i)
            {
                accum.y = bdata[i] = (bdata[i] - accum.x*c_Ninf - accum.y)*c_Linf2;
                swap(accum.x,accum.y);
            }
        }

        m -= ty;
    }

    __syncthreads();

#pragma unroll
    for(int i=0; i<WS-2; i+=SOW)
    {
        g_inout[i*c_width] = block[ty+i][tx];
        g_inout[(i+WS)*c_width] = block[ty+i+WS][tx];
    }

    if(ty < 2)
    {
        g_inout[(WS-2)*c_width] = block[ty+WS-2][tx];
        g_inout[(WS-2+WS)*c_width] = block[ty+WS-2+WS][tx];
    }

    if(ty < 2)
    {
        m += ty;

        float (*bdata)[WS+1] = (float (*)[WS+1]) &block[ty*WS][tx];

        int outidx = m*c_width + n*WS + tx; 
        float2 accum;

        // first block
        if(m < c_m_size-1)
        {
            accum.x = bdata[0][0];
            accum.y = bdata[1][0] -= c_Minf*accum.x;

#pragma unroll
            for(int i=2; i<WS; ++i)
            {
                accum.x = bdata[i][0] -= c_Minf*accum.y + c_Linf2*accum.x;
                swap(accum.x, accum.y);
            }
        }
        else
        {
            accum.x = bdata[0][0];
            accum.y = bdata[1][0] -= c_Minf*accum.x;

            for(int i=2; i<WS-1; ++i)
            {
                accum.x = bdata[i][0] -= c_Minf*accum.y + c_Linf2*accum.x;
                swap(accum.x, accum.y);
            }
            accum.x = bdata[WS-1][0] -= c_Minf*accum.y + c_Linf2*accum.x;
            swap(accum.x, accum.y);
        }

        g_ubar[outidx] = accum;

        if(m < c_m_size-1)
        {
            accum.y = bdata[WS-1][0] * c_Linf2;
            accum.x = (bdata[WS-2][0] - accum.y*c_Ninf)*c_Linf2;

#pragma unroll
            for(int i=WS-3; i>=0; --i)
            {
                accum.y = (bdata[i][0] - accum.x*c_Ninf - accum.y)*c_Linf2;
                swap(accum.x,accum.y);
            }
        }
        else
        {
            int i = WS-1;

            accum.y = bdata[i--][0] * c_Linf2;
            accum.x = (bdata[i--][0] - accum.y*c_Ninf)*c_Linf2;
            for(; i>=0; --i)
            {
                accum.y = (bdata[i][0] - accum.x*c_Ninf - accum.y)*c_Linf2;
                swap(accum.x,accum.y);
            }
        }

        g_vhat[outidx] = accum;
    }
}

__global__ __launch_bounds__(WS*SOW, DNB)
void alg4_stage7( float *g_inout,
                  const float2 *g_u,
                  const float2 *g_v )
{
    int m = blockIdx.y, n = blockIdx.x;
    int tx = threadIdx.x, ty = threadIdx.y;

    __shared__ float block[WS][WS+1];

    g_inout += (m*WS + ty)*c_width + n*WS + tx;

#pragma unroll
    for(int i=0; i<WS-2; i+=SOW)
    {
        block[tx][ty+i] = g_inout[i*c_width];
    }

    if(ty < 2)
    {
        block[tx][ty+WS-2] = g_inout[(WS-2)*c_width];
    }

    __syncthreads();

    if(ty == 0)
    {
        int outidx = m*c_width + n*WS + tx; 

        float *bdata = block[tx];

        // first block
        if(m < c_m_size-1)
        {
            float2 accum;

            if(m == 0)
            {
                accum.x = bdata[0];
                accum.y = bdata[1] -= c_Minf*accum.x;
            }
            else
                accum = g_u[outidx-c_width];

#pragma unroll
            for(int i=0; i<WS; ++i)
            {
                accum.x = bdata[i] -= c_Minf*accum.y + c_Linf2*accum.x;
                swap(accum.x, accum.y);
            }
        }
        else
        {
            float2 accum  = g_u[outidx-c_width];

            for(int i=0; i<WS-1; ++i)
            {
                accum.x = bdata[i] -= c_Minf*accum.y + c_Linf2*accum.x;
                swap(accum.x, accum.y);
            }
            bdata[WS-1] -= c_Minf*accum.y + c_Linf2*accum.x;
        }

        if(m < c_m_size-1)
        {
            float2 accum = g_v[outidx+c_width];

#pragma unroll
            for(int i=WS-1; i>=0; --i)
            {
                bdata[i] = accum.y = (bdata[i] - accum.x*c_Ninf - accum.y)*c_Linf2;
                swap(accum.x,accum.y);
            }
        }
        // last block
        else
        {
            int i = WS-1;

            float2 accum;
            accum.y = bdata[i--] *= c_Linf2;
            accum.x = bdata[i] = (bdata[i] - accum.y*c_Ninf)*c_Linf2;
            --i;
            for(; i>=0; --i)
            {
                bdata[i] = accum.y = (bdata[i] - accum.x*c_Ninf - accum.y)*c_Linf2;
                swap(accum.x,accum.y);
            }
        }
    }

    __syncthreads();

#pragma unroll
    for(int i=0; i<WS-2; i+=SOW)
    {
        g_inout[i*c_width] = block[tx][ty+i] * c_iR2;
    }

    if(ty < 2)
    {
        g_inout[(WS-2)*c_width] = block[tx][ty+WS-2] * c_iR2;
    }
}

//-- Algorithm 5_1 ------------------------------------------------------------

__global__ __launch_bounds__(WS*SOW, MBO)
void alg5_stage1( const float *g_in,
                  float *g_transp_pybar,
                  float *g_transp_ezhat,
                  float *g_ptucheck,
                  float *g_etvtilde )
{
    int tx = threadIdx.x, ty = threadIdx.y, m = blockIdx.x, n = blockIdx.y;
    __shared__ float block[WS][WS+1];

    // make g_in point to the data we'll work with
    g_in += (n*WS+ty)*c_width + m*WS+tx;

    float (*bdata)[WS+1] = (float (*)[WS+1]) &block[ty][tx];

    // load data into shared memory
    int i;
#pragma unroll
    for(i=0; i<WS-(WS%SOW); i+=SOW)
    {
        **bdata = *g_in;
        bdata += SOW;
        g_in += SOW*c_width;
    }

    if(ty < WS%SOW)
    {
        **bdata = *g_in;
    }

    // We use a transposed matrix for pybar and ezhat to have
    // coalesced memory accesses. This is the movement for these
    // transposed buffers.
    g_transp_pybar += m*c_height + n*WS + tx; 
    g_transp_ezhat += m*c_height + n*WS + tx;
    g_ptucheck += n*c_width + m*WS + tx;
    g_etvtilde += n*c_width + m*WS + tx;

    __syncthreads();

    float prev;

    if(ty == 0)
    {
        // scan columns
        {
            float *bdata = block[tx];

            // calculate pybar, scan left -> right

            prev = *bdata++;

#pragma unroll
            for(int j=1; j<WS; ++j, ++bdata)
                prev = *bdata -= prev*c_a1;

            *g_transp_pybar = prev*c_b0;
            
            // calculate ezhat, scan right -> left

            prev = *--bdata;
            --bdata;

#pragma unroll
            for(int j=WS-2; j>=0; --j, --bdata)
                prev = *bdata -= prev*c_a1;

            *g_transp_ezhat = prev*c_b0*c_b0;
        }

        // scan rows
        {
            float (*bdata)[WS+1] = (float (*)[WS+1]) &block[0][tx];

            // calculate ptucheck, scan top -> down

            prev = **bdata++;

#pragma unroll
            for(int i=1; i<WS; ++i, ++bdata)
                prev = **bdata -= prev*c_a1;

            *g_ptucheck = prev*c_b0*c_b0*c_b0;

            // calculate etvtilde, scan bottom -> up

            prev = **--bdata;
            --bdata;

            for(int i=WS-2; i>=0; --i, --bdata)
                prev = **bdata - prev*c_a1;

            *g_etvtilde = prev*c_b0*c_b0*c_b0*c_b0;
        }
    }
}

__global__ __launch_bounds__(WS*DW, DNB)
void alg5_stage2_3( float *g_transp_pybar,
                    float *g_transp_ezhat )
{
    int tx = threadIdx.x, ty = threadIdx.y, n = blockIdx.y;

    __shared__ float transp_block[DW][WS];
    float *bdata = &transp_block[ty][tx];

    // P(ybar) -> P(y) processing --------------------------------------

    float *transp_pybar = g_transp_pybar + ty*c_height + n*WS+tx;

    // first column-transp_block

    // read P(ybar)
    *bdata = *transp_pybar;

    float py; // P(Y)

    __syncthreads();

    if(ty == 0)
    {
        float (*bdata)[WS] = (float (*)[WS]) &transp_block[0][tx];

        // (24): P_m(y) = P_m(ybar) + A^b_F * P_{m-1}(y)
        py = **bdata++;

#pragma unroll
        for(int m=1; m<blockDim.y; ++m, ++bdata)
            **bdata = py = **bdata + c_AbF*py;
    }

    __syncthreads();

    // write P(y)
    if(ty > 0) // first one doesn't need fixing
        *transp_pybar = *bdata;

    transp_pybar += c_height*blockDim.y;

    // middle column-blocks

    int m = blockDim.y;
    if(m == DW)
    {
        for(; m<c_m_size-(c_m_size%DW); m+=DW)
        {
            *bdata = *transp_pybar;

            __syncthreads();

            if(ty == 0)
            {
                float (*bdata)[WS] = (float (*)[WS]) &transp_block[0][tx];
#pragma unroll
                for(int dm=0; dm<DW; ++dm, ++bdata)
                    **bdata = py = **bdata + c_AbF*py;
            }

            __syncthreads();

            *transp_pybar = *bdata;
            transp_pybar += c_height*DW;
        }
    }

    // remaining column-transp_blocks

    if(m < c_m_size)
    {
        int remaining = c_m_size - m;

        if(remaining > 0)
            *bdata = *transp_pybar;

        __syncthreads();

        if(ty == 0)
        {
            float (*bdata)[WS] = (float (*)[WS]) &transp_block[0][tx];
#pragma unroll
            for(int dm=0; dm<remaining; ++dm, ++bdata)
                **bdata = py = **bdata + c_AbF*py;

        }

        __syncthreads();

        if(remaining > 0)
            *transp_pybar = *bdata;
    }

    // E(zhat) -> E(z) processing --------------------------------------

    int idx = (c_m_size-1-ty)*c_height + n*WS+tx;

    const float *transp_pm1y = g_transp_pybar + idx - c_height;

    // last column-transp_block

    float *transp_ezhat = g_transp_ezhat + idx;

    // all pybars must be updated!
    __syncthreads();

    float ez;

    {
        *bdata = *transp_ezhat;

        if(m-ty > 0)
            *bdata += *transp_pm1y*c_HARB_AFP;

        __syncthreads();

        if(ty == 0)
        {
            float (*bdata)[WS] = (float (*)[WS]) &transp_block[0][tx];
            ez = **bdata++;

#pragma unroll
            for(int dm=1; dm<blockDim.y; ++dm, ++bdata)
                **bdata = ez = **bdata + c_AbR*ez;
        }

        __syncthreads();

        *transp_ezhat = *bdata;

        transp_ezhat -= c_height*blockDim.y;
        transp_pm1y -= c_height*blockDim.y;
    }

    // middle column-transp_blocks
    m = c_m_size-1 - blockDim.y;
    if(blockDim.y == DW)
    {
        for(; m>=c_m_size%DW; m-=DW)
        {
            *bdata = *transp_ezhat;

            if(m-ty > 0)
                *bdata += *transp_pm1y*c_HARB_AFP;

            __syncthreads();

            if(ty == 0)
            {
                float (*bdata)[WS] = (float (*)[WS]) &transp_block[0][tx];
#pragma unroll
                for(int dm=0; dm<DW; ++dm, ++bdata)
                    **bdata = ez = **bdata + c_AbR*ez;
            }

            __syncthreads();

            *transp_ezhat = *bdata;

            transp_ezhat -= DW*c_height;
            transp_pm1y -= DW*c_height;
        }
    }

    // remaining column-blocks

    if(m >= 0)
    {
        int remaining = m+1;

        if(m-ty >= 0)
        {
            *bdata = *transp_ezhat;
        
            if(m-ty > 0)
                *bdata += *transp_pm1y*c_HARB_AFP;
        }

        __syncthreads();

        if(ty == 0)
        {
            float (*bdata)[WS] = (float (*)[WS]) &transp_block[0][tx];
            // (24): P_m(y) = P_m(ybar) + A^b_F * P_{m-1}(y)
#pragma unroll
            for(int dm=0; dm<remaining; ++dm, ++bdata)
                **bdata = ez = **bdata + c_AbR*ez;
        }

        __syncthreads();

        if(m-ty >= 0)
            *transp_ezhat = *bdata;
    }
}

__global__ __launch_bounds__(WS*CFW, ONB)
void alg5_stage4_5( float *g_ptucheck,
                    float *g_etvtilde,
                    const float *g_transp_py,
                    const float *g_transp_ez )
{
    int tx = threadIdx.x, ty = threadIdx.y, m = blockIdx.x;

    __shared__ float block[CFW][WS];
    float *bdata = &block[ty][tx];

    // P(ucheck) -> P(u) processing --------------------------------------

	volatile __shared__ float block_RD_raw[CFW][16+32+1];
	volatile float (*block_RD)[16+32+1] 
        = (float (*)[16+32+1]) &block_RD_raw[0][16];
    if(ty < CFW)
        block_RD_raw[ty][tx] = 0;

#define CALC_DOT(RES, V1, V2) \
    block_RD[ty][tx] = V1*V2; \
    block_RD[ty][tx] += block_RD[ty][tx-1]; \
    block_RD[ty][tx] += block_RD[ty][tx-2]; \
    block_RD[ty][tx] += block_RD[ty][tx-4]; \
    block_RD[ty][tx] += block_RD[ty][tx-8]; \
    block_RD[ty][tx] += block_RD[ty][tx-16]; \
    float RES = block_RD[ty][31];

    float *ptucheck = g_ptucheck + m*WS+tx + ty*c_width;

    // first row-block

    int idx = m*c_height + ty*WS+tx;

    const float *transp_pm1ybar = g_transp_py + idx - c_height,
                *transp_em1zhat = g_transp_ez + idx + c_height;

    float ptu;

    {
        // read P(ucheck)
        *bdata = *ptucheck;

        if(m > 0)
        {
            CALC_DOT(dot, *transp_pm1ybar, c_TAFB[tx]);
            *bdata += dot*c_ARB_AFP_T[tx];
        }

        if(m < c_m_size-1)
        {
            CALC_DOT(dot, *transp_em1zhat, c_TAFB[tx]);
            *bdata += dot*c_ARE_T[tx];
        }

        transp_pm1ybar += WS*blockDim.y;
        transp_em1zhat += WS*blockDim.y;

        __syncthreads();

        if(ty == 0)
        {
            float (*bdata2)[WS] = (float (*)[WS]) &block[0][tx];

            ptu = **bdata2++;

#pragma unroll
            for(int n=1; n<blockDim.y; ++n, ++bdata2)
                **bdata2 = ptu = **bdata2 + c_AbF*ptu;
        }

        __syncthreads();

        // write P(u)
        *ptucheck = *bdata;

        ptucheck += blockDim.y*c_width;
    }

    // middle row-blocks

    int n = blockDim.y;
    if(n == CFW)
    {
        int nmax = c_n_size-(c_n_size%CFW);
        for(; n<nmax; n+=CFW)
        {
            *bdata = *ptucheck;

            if(m > 0)
            {
                CALC_DOT(dot, *transp_pm1ybar, c_TAFB[tx]);
                *bdata += dot*c_ARB_AFP_T[tx];
            }

            if(m < c_m_size-1)
            {
                CALC_DOT(dot, *transp_em1zhat, c_TAFB[tx]);
                *bdata += dot*c_ARE_T[tx];
            }

            transp_pm1ybar += WS*CFW;
            transp_em1zhat += WS*CFW;

            __syncthreads();

            if(ty == 0)
            {
                float (*bdata2)[WS] = (float (*)[WS]) &block[0][tx];

#pragma unroll
                for(int dn=0; dn<CFW; ++dn, ++bdata2)
                    **bdata2 = ptu = **bdata2 + c_AbF*ptu;
            }

            __syncthreads();

            *ptucheck = *bdata;

            ptucheck += CFW*c_width;

        }
    }

    // remaining row-blocks

    if(n < c_n_size)
    {

        if(n+ty < c_n_size)
        {
            *bdata = *ptucheck;

            if(m > 0)
            {
                CALC_DOT(dot, *transp_pm1ybar, c_TAFB[tx]);
                *bdata += dot*c_ARB_AFP_T[tx];
            }

            if(m < c_m_size-1)
            {
                CALC_DOT(dot, *transp_em1zhat, c_TAFB[tx]);
                *bdata += dot*c_ARE_T[tx];
            }
        }

        int remaining = c_n_size-n;
        __syncthreads();

        if(ty == 0)
        {
            float (*bdata2)[WS] = (float (*)[WS]) &block[0][tx];
#pragma unroll
            for(int dn=0; dn<remaining; ++dn, ++bdata2)
                **bdata2 = ptu = **bdata2 + c_AbF*ptu;
        }

        __syncthreads();

        if(n+ty < c_n_size)
            *ptucheck = *bdata;
    }

    // E(utilde) -> E(u) processing --------------------------------------

    // last row-block

    idx = (c_n_size-1-ty)*c_width + m*WS+tx;
    int transp_idx = m*c_height + (c_n_size-1-ty)*WS+tx;

    float *etvtilde = g_etvtilde + idx;

    transp_pm1ybar = g_transp_py + transp_idx-c_height;
    transp_em1zhat = g_transp_ez + transp_idx+c_height;

    const float *ptmn1u = g_ptucheck + idx - c_width;

    // all ptuchecks must be updated!
    __syncthreads();

    float etv;

    n = c_n_size-1;

    {
        block[ty][tx] = *etvtilde;


        if(m > 0)
        {
            CALC_DOT(dot, *transp_pm1ybar, c_HARB_AFB[tx]);
            *bdata += dot*c_ARB_AFP_T[tx];
        }

        if(m < c_m_size-1)
        {
            CALC_DOT(dot, *transp_em1zhat, c_HARB_AFB[tx]);
            *bdata += dot*c_ARE_T[tx];
        }

        if(n-ty > 0)
            *bdata += *ptmn1u*c_HARB_AFP;

        transp_pm1ybar -= WS*blockDim.y;
        transp_em1zhat -= WS*blockDim.y;
        ptmn1u -= c_width*blockDim.y;

        __syncthreads();

        if(ty == 0)
        {
            float (*bdata2)[WS] = (float (*)[WS]) &block[0][tx];

            etv = **bdata2++;

#pragma unroll
            for(int dn=1; dn<blockDim.y; ++dn, ++bdata2)
                **bdata2 = etv = **bdata2 + c_AbR*etv;
        }

        __syncthreads();

        *etvtilde = *bdata;

        etvtilde -= c_width*blockDim.y;

        n -= blockDim.y;
    }

    // middle row-blocks
    if(blockDim.y == CFW)
    {
        int nmin = c_n_size%CFW;
        for(; n>=nmin; n-=CFW)
        {

            *bdata = *etvtilde;


            if(m > 0)
            {
                CALC_DOT(dot, *transp_pm1ybar, c_HARB_AFB[tx]);
                *bdata += dot*c_ARB_AFP_T[tx];
            }

            if(m < c_m_size-1)
            {
                CALC_DOT(dot, *transp_em1zhat, c_HARB_AFB[tx]);
                *bdata += dot*c_ARE_T[tx];
            }

            if(n-ty > 0)
                *bdata += *ptmn1u*c_HARB_AFP;

            transp_pm1ybar -= WS*CFW;
            transp_em1zhat -= WS*CFW;
            ptmn1u -= CFW*c_width;

            __syncthreads();

            if(ty == 0)
            {
                float (*bdata2)[WS] = (float (*)[WS]) &block[0][tx];
#pragma unroll
                for(int dn=0; dn<CFW; ++dn, ++bdata2)
                    **bdata2 = etv = **bdata2 + c_AbR*etv;
            }

            __syncthreads();

            *etvtilde = *bdata;

            etvtilde -= CFW*c_width;
        }
    }

    // remaining row-blocks

    if(n >= 0)
    {

        if(n-ty >= 0)
        {
            *bdata = *etvtilde;
            if(n-ty > 0)
                *bdata += *ptmn1u*c_HARB_AFP;

            if(m > 0)
            {
                CALC_DOT(dot, *transp_pm1ybar, c_HARB_AFB[tx]);
                *bdata += dot*c_ARB_AFP_T[tx];
            }

            if(m < c_m_size-1)
            {
                CALC_DOT(dot, *transp_em1zhat, c_HARB_AFB[tx]);
                *bdata += dot*c_ARE_T[tx];
            }
        }

        int remaining = n+1;
        __syncthreads();

        if(ty == 0)
        {
            float (*bdata2)[WS] = (float (*)[WS]) &block[0][tx];
#pragma unroll
            for(int dn=0; dn<remaining; ++dn, ++bdata2)
                **bdata2 = etv = **bdata2 + c_AbR*etv;
        }

        __syncthreads();

        if(n-ty >= 0)
            *etvtilde = *bdata;
    }
#undef CALC_DOT
}

__global__ __launch_bounds__(WS*SOW, MBO)
void alg5_stage6( float *g_inout,
                  const float *g_transp_py,
                  const float *g_transp_ez,
                  const float *g_ptu,
                  const float *g_etv )
{
    int tx = threadIdx.x, ty = threadIdx.y, m = blockIdx.x, n = blockIdx.y;

    __shared__ float block[WS][WS+1];

    const float *in = g_inout + (n*WS+ty)*c_width + m*WS+tx;

    float (*curb)[WS+1] = (float (*)[WS+1]) &block[ty][tx];

    // load data into shared memory
    int i;
#pragma unroll
    for(i=0; i<WS-(WS%SOW); i+=SOW)
    {
        **curb = *in;
        in += SOW*c_width;
        curb += SOW;
    }

    if(ty < WS%SOW)
    {
        **curb = *in;
    }

    __shared__ float py[WS], ez[WS], ptu[WS], etv[WS];

    switch(ty)
    {
    case 0:
        if(m > 0)
            py[tx] = g_transp_py[(n*WS + tx) + (m-1)*c_height] / c_b0;
        else
            py[tx] = 0;
        break;
    case 1:
        if(m < c_m_size-1)
            ez[tx] = g_transp_ez[(n*WS + tx) + (m+1)*c_height];
        else
            ez[tx] = 0;
        break;
    case 2:
        if(n > 0)
            ptu[tx] = g_ptu[(m*WS + tx) + (n-1)*c_width] / c_b0;
        else
            ptu[tx] = 0;
        break;
    case 3:
        if(n < c_n_size-1)
            etv[tx] = g_etv[(m*WS + tx) + (n+1)*c_width];
        else
            etv[tx] = 0;
        break;
    }

    __syncthreads();

    if(ty == 0)
    {
        float b0_2 = c_b0*c_b0;
        {
            float *bdata = block[tx];

            // calculate y ---------------------

            float prev = py[tx];

#pragma unroll
            for(int j=0; j<WS; ++j, ++bdata)
                *bdata = prev = *bdata - prev*c_a1;

            // calculate z ---------------------

            prev = ez[tx];

            --bdata;
            for(int j=WS-1; j>=0; --j, --bdata)
                *bdata = prev = *bdata*b0_2 - prev*c_a1;
        }

        {
            float (*bdata)[WS+1] = (float (*)[WS+1]) &block[0][tx];

            // calculate u ---------------------

            float prev = ptu[tx];

#pragma unroll
            for(int i=0; i<WS; ++i, ++bdata)
                **bdata = prev = **bdata - prev*c_a1;

            // calculate v ---------------------
            float *out = g_inout + ((n+1)*WS-1)*c_width + m*WS+tx;

            prev = etv[tx];

            --bdata;
            for(int i=WS-1; i>=0; --i) 
            {
                *out = prev = **bdata-- *b0_2 - prev*c_a1;
                out -= c_width;
            }
        }
    }
}

//-- Host ---------------------------------------------------------------------

__host__
void alg4( float *inout,
           const int& h,
           const int& w,
           const float& b0,
           const float& a1,
           const float& a2 )
{
    up_constants_coefficients2( b0, a1, a2 );

    dim3 cg_img;
    up_constants_sizes( cg_img, h, w );

    dvector<float> d_img(inout, h*w);

    dvector<float2> d_transp_ybar(cg_img.y*w),
        d_transp_zhat(cg_img.y*w),
        d_ubar(cg_img.x*h),
        d_vhat(cg_img.x*h);

    dvector<float2> d_transp_y, d_transp_z, d_u, d_v;

    alg4_stage1<<< cg_img, dim3(WS, SOW) >>>(
        d_img, d_transp_ybar, d_transp_zhat );

    alg4_stage2_3_or_5_6<<< dim3((h+MTS-1)/MTS, 1), dim3(MTS, 1) >>>(
        d_transp_ybar, d_transp_zhat );

    swap( d_transp_ybar, d_transp_y );
    swap( d_transp_zhat, d_transp_z );

    alg4_stage4<<< dim3(cg_img.x, (cg_img.y+2-1)/2), dim3(WS, SOW) >>>(
        d_img, d_transp_y, d_transp_z, d_ubar, d_vhat );

    alg4_stage2_3_or_5_6<<< dim3((w+MTS-1)/MTS, 1), dim3(MTS, 1) >>>(
        d_ubar, d_vhat );

    swap( d_ubar, d_u );
    swap( d_vhat, d_v );

    alg4_stage7<<< cg_img, dim3(WS, SOW) >>>(
        d_img, d_u, d_v );

    d_img.copy_to(inout, h*w);
}

__host__
void alg5( float *inout,
           const int& h,
           const int& w,
           const float& b0,
           const float& a1 )
{
    up_constants_coefficients1( b0, a1 );

    dim3 cg_img;
    up_constants_sizes( cg_img, h, w );

    dvector<float> d_img(inout, h*w);

    dvector<float> d_transp_pybar(cg_img.x*h),
        d_transp_ezhat(cg_img.x*h),
        d_ptucheck(cg_img.y*w),
        d_etvtilde(cg_img.y*w);
                   
    dvector<float> d_transp_py, d_transp_ez, d_ptu, d_etv;

    alg5_stage1<<< cg_img, dim3(WS, SOW) >>>(
        d_img, d_transp_pybar, d_transp_ezhat, d_ptucheck, d_etvtilde );

    alg5_stage2_3<<< dim3(1, cg_img.y), dim3(WS, std::min<int>(cg_img.x, DW)) >>>(
        d_transp_pybar, d_transp_ezhat );

    swap(d_transp_pybar, d_transp_py);
    swap(d_transp_ezhat, d_transp_ez);

    alg5_stage4_5<<< dim3(cg_img.x, 1), dim3(WS, std::min<int>(cg_img.y, CFW)) >>>(
        d_ptucheck, d_etvtilde, d_transp_py, d_transp_ez );

    swap(d_ptucheck, d_ptu);
    swap(d_etvtilde, d_etv);

    alg5_stage6<<< cg_img, dim3(WS, SOW) >>>(
        d_img, d_transp_py, d_transp_ez, d_ptu, d_etv );

    d_img.copy_to(inout, h*w);
}

__host__
void gaussian_gpu( float **inout,
                   const int& h,
                   const int& w,
                   const int& d,
                   const float& s )
{
    float b10, a11, b20, a21, a22;
    weights1( s, b10, a11 );
    weights2( s, b20, a21, a22 );
    for (int c = 0; c < d; c++) {
        alg5( inout[c], h, w, b10, a11 );
        alg4( inout[c], h, w, b20, a21, a22 );
    }
}

__host__
void gaussian_gpu( float *inout,
                   const int& h,
                   const int& w,
                   const float& s )
{
    float b10, a11, b20, a21, a22;
    weights1( s, b10, a11 );
    weights2( s, b20, a21, a22 );
    alg5( inout, h, w, b10, a11 );
    alg4( inout, h, w, b20, a21, a22 );
}

__host__
void bspline3i_gpu( float **inout,
                    const int& h,
                    const int& w,
                    const int& d )
{
    const float alpha = 2.f - sqrt(3.f);
    for (int c = 0; c < d; c++) {
        alg5( inout[c], h, w, 1.f+alpha, alpha );
    }
}

__host__
void bspline3i_gpu( float *inout,
                    const int& h,
                    const int& w )
{
    const float alpha = 2.f - sqrt(3.f);
    alg5( inout, h, w, 1.f+alpha, alpha );
}

//=============================================================================
} // namespace gpufilter
//=============================================================================
// vi: ai ts=4 sw=4
